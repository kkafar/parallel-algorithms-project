
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <numeric>
#include <ranges>
#include <cmath>

__global__ void langevin_equation(float *output, int n, float gamma, unsigned long long seed) {
    int idx = blockIdx.x;
    float x = 0.0;
    float v = 0.0;
    float t = 0.0;
    float dt = 0.001;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    while (true) {
        v = v + (-gamma * v * dt + hiprand_normal(&state));
        x = x + v * dt;
        t += dt;

        if ((x >= 0 ? x : -x ) > 1) {
            break;
        }
    }

    output[idx] = t;
}

int main() {
    float gamma = 0.33;

    // Set the size of the array
    int n = 24;

    // Allocate memory on the host
    float *h_output = (float*)malloc(n * sizeof(float));

    // Allocate memory on the device
    float *d_output;
    hipMalloc((void**)&d_output, n * sizeof(float));

    // Set mean, standard deviation, and seed
    unsigned long long seed = 1234;

    // Launch the CUDA kernel
    int blockSize = 1;
    int numBlocks = (n + blockSize - 1) / blockSize;

    langevin_equation<<<numBlocks, blockSize>>>(d_output, n, gamma, seed);

    // Copy the results back to the host
    hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    const float total_avg = std::accumulate(h_output, h_output + n, 0.0f) / static_cast<float>(n);
    const auto square_fn = [total_avg](auto val) { return (val - total_avg) * (val - total_avg); };
    const float stddev = std::sqrt(std::transform_reduce(h_output, h_output + n, 0.0f, std::plus{}, square_fn) / (n - 1));

    for (int i = 0; i < n; ++i) {
        printf("%f ", h_output[i]);
    }
    printf("\n");
    printf("Avg: %.2f with std: %.2f\n", total_avg, stddev);

    // Free device and host memory
    hipFree(d_output);
    free(h_output);

    return 0;
}

